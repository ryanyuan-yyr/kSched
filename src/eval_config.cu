#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <unistd.h>

#include <algorithm>
#include <fstream>
#include <tuple>
#include <utility>

#include "ksched.cuh"

constexpr int nrepeat = 3;

__host__ int main() {
  // set up max connectioin
  const char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
  setenv(iname, "32", 1);

  Kernel vec_add_kernel{"build/matrix_mul.so"};
  Kernel sqrt_pow_kernel{"build/matrix_transpose.so"};

  constexpr int NSTREAM = 2;
  hipStream_t streams[NSTREAM];
  for (size_t i = 0; i < NSTREAM; i++) {
    hipStreamCreate(streams + i);
  }

  CoSchedKernels co_kernel{vec_add_kernel, sqrt_pow_kernel, streams[0],
                           streams[1]};

  printf("Boundary %d, %d\n", co_kernel.get_boundary().first,
         co_kernel.get_boundary().second);

  // warmup
  printf("============================== Warm up Duration %lf\n",
         co_kernel.eval_cosched_time(co_kernel.get_boundary(), nrepeat));

  // Serial
  printf("============================== Not sliced Duration %lf\n",
         co_kernel.eval_cosched_time(co_kernel.get_boundary(), nrepeat));

  // Mix
  Config granularity{co_kernel.get_granularity()};
  printf("Granularity %d, %d\n", granularity.first, granularity.second);
  auto subregion = std::pair<Axes<int>, Axes<int>>{
      {granularity.first * 1, granularity.second * 1},
      {granularity.first * 49, granularity.second * 97}};
  printf("Subregion (%d, %d), (%d, %d)\n", subregion.first.first,
         subregion.first.second, subregion.second.first,
         subregion.second.second);

  std::ofstream output{"data/comprehensive_tune_config"};

  for (int i = subregion.first.first; i < subregion.second.first;
       i += granularity.first) {
    for (int j = subregion.first.second; j < subregion.second.second;
         j += granularity.second) {
      output << co_kernel.eval_cosched_time({i, j}, nrepeat, false, false,
                                            false)
             << " ";
    }
    output << "\n";
  }
}